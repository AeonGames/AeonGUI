#include "hip/hip_runtime.h"
#include <stdio.h>
#include "cuda_functions.h"

namespace AeonGUI
{
__global__ void nearest()
{
    printf("nearest %u\n",threadIdx.x);
}

void NearestNeighbour()
{
    nearest<<<16,1>>>();
    hipDeviceSynchronize();
}
}
